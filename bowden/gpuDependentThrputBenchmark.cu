#include "hip/hip_runtime.h"
/*
 * Author: Brian Bowden
 * Date: 4/15/12
 *
 * gpuDependentThrputBenchmark.cu
 *
 */

#include <stdlib.h>
#include <stdio.h>
#include "repeat.h"

#define REPEAT(iters)	repeat ## iters

#define INT 0
#define UINT 1
#define FLOAT 2
#define DOUBLE 3

// ############# CHANGE BELOW 2 LINES FOR DIFFERENT DATA TYPES!!!!! ######################
typedef float TYPE;
#define DATATYPE (FLOAT)
// ############# 1 FOR MAXIMUM OCCUPANCY, 0 FOR VARYING WARPS #######################
#define MAX_OCCUPANCY 0

int total_threads;
const int threads_per_warp = 32;
const int max_warps = 48;
const int number_multi_processors = 14;
const float clock_speed = 1.15e9;	//1.15 GHz
const int number_runs = 25;
const int N = threads_per_warp * max_warps * number_multi_processors;
TYPE operand = 10;

const int block_size = 2;
TYPE* host_A;
TYPE* host_B;
TYPE* device_A;
TYPE* device_B;
	
hipEvent_t start, stop;

void print_results(double average_time, int number_runs, int total_threads, int iterations) 
{
	int number_instructions = total_threads * iterations * 2;
	average_time /= (double) number_runs;
	long long int number_cycles = (long long int) ((average_time * clock_speed) / 1000);
	double throughput = (double) number_instructions / number_cycles;
	
#if (MAX_OCCUPANCY)
	printf("Average Time for %d iterations : %g (ms)\n", iterations, average_time);
	printf("Total number cycles : %ld\n", number_cycles);
	printf("Throughput : %0.3g\n", throughput);
#endif
#if (!MAX_OCCUPANCY)
	int number_warps = total_threads / threads_per_warp;
	printf("%d warps : %0.3g\n", number_warps, throughput);
#endif
}

__global__ void kernelAdd(TYPE* A, TYPE* B)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	TYPE a_val = A[i];
	TYPE b_val = B[i];
	repeat4096(a_val += b_val; b_val += a_val;);
	A[i] = a_val;
	B[i] = b_val;
}

void getAddThroughput()
{
	double average_time = 0.0;
	float time_elapsed;
	int number_threads = 0;
	int iterations = 4096;
	if (total_threads == 32 || MAX_OCCUPANCY) printf("\n---------------Addition--------------------\n"); 
	number_threads = total_threads / block_size;
	for (int j = 0; j < number_runs; j++) 
	{
		hipEventRecord(start, 0);
		kernelAdd<<<block_size * number_multi_processors, number_threads>>>(device_A, device_B);
		hipEventRecord(stop, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_elapsed, start, stop);
		average_time += time_elapsed;
	}
	print_results(average_time, number_runs, total_threads, iterations);
}

__global__ void kernelSub(TYPE* A, TYPE* B)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	TYPE a_val = A[i];
	TYPE b_val = B[i];
	repeat4096(a_val -= b_val; b_val -= a_val;);
	A[i] = a_val;
	B[i] = b_val;
}

void getSubtractThroughput()
{
	double average_time = 0.0;
	float time_elapsed;
	int number_threads = 0;
	int iterations = 4096;
	if (total_threads == 32 || MAX_OCCUPANCY) printf("\n--------------Subtraction------------------\n"); 
	number_threads = total_threads / block_size;
	for (int j = 0; j < number_runs; j++) 
	{
		hipEventRecord(start, 0);
		kernelSub<<<block_size * number_multi_processors, number_threads>>>(device_A, device_B);
		hipEventRecord(stop, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_elapsed, start, stop);
		average_time += time_elapsed;
	}
	print_results(average_time, number_runs, total_threads, iterations);
}


__global__ void kernelMul(TYPE* A, TYPE* B)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	TYPE a_val = A[i];
	TYPE b_val = B[i];
	repeat4096(a_val *= b_val; b_val *= a_val;);	
	A[i] = a_val;
	B[i] = b_val;
}

void getMultiplyThroughput()
{
	double average_time = 0.0;
	float time_elapsed;
	int number_threads = 0;
	int iterations = 4096;
	if (total_threads == 32 || MAX_OCCUPANCY) printf("\n------------Multiplication-----------------\n"); 
	number_threads = total_threads / block_size;
	for (int j = 0; j < number_runs; j++) 
	{
		hipEventRecord(start, 0);
		kernelMul<<<block_size * number_multi_processors, number_threads>>>(device_A, device_B);
		hipEventRecord(stop, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_elapsed, start, stop);
		average_time += time_elapsed;
	}
	print_results(average_time, number_runs, total_threads, iterations);
}


__global__ void kernelDiv(TYPE* A, TYPE* B)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	TYPE a_val = A[i];
	TYPE b_val = B[i];
	repeat512(a_val /= b_val; b_val /= a_val;);	
	A[i] = a_val;
	B[i] = b_val;
}

void getDivideThroughput()
{
	double average_time = 0.0;
	float time_elapsed;
	int number_threads = 0;
	int iterations = 512;
	if (total_threads == 32 || MAX_OCCUPANCY) printf("\n---------------Division--------------------\n"); 
	number_threads = total_threads / block_size;
	for (int j = 0; j < number_runs; j++) 
	{
		hipEventRecord(start, 0);
		kernelDiv<<<block_size * number_multi_processors, number_threads>>>(device_A, device_B);
		hipEventRecord(stop, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_elapsed, start, stop);
		average_time += time_elapsed;
	}
	print_results(average_time, number_runs, total_threads, iterations);
}

__global__ void kernelMAD(TYPE* A, TYPE* B)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	TYPE a_val = A[i];
	TYPE b_val = B[i];
	repeat4096(a_val *= b_val; a_val += b_val; b_val *= a_val; b_val += a_val;);
	A[i] = a_val;
	B[i] = b_val;
}

void getMultiplyAddThroughput()
{
	double average_time = 0.0;
	float time_elapsed;
	int number_threads = 0;
	int iterations = 4096;
	if (total_threads == 32 || MAX_OCCUPANCY) printf("\n-------------Multiply-Add------------------\n"); 
	number_threads = total_threads / block_size;
	for (int j = 0; j < number_runs; j++) 
	{
		hipEventRecord(start, 0);
		kernelMAD<<<block_size * number_multi_processors, number_threads>>>(device_A, device_B);
		hipEventRecord(stop, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_elapsed, start, stop);
		average_time += time_elapsed;
	}
	print_results(average_time, number_runs, total_threads, iterations);
}


#if DATATYPE == INT || DATATYPE == UINT
__global__ void kernelVectorAdd(TYPE* A, TYPE* B, TYPE operand)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	TYPE a_val = A[i];
	TYPE b_val = B[i];
	repeat2048(a_val += b_val + operand; b_val += a_val + operand;);
	A[i] = a_val;
	B[i] = b_val;
}

void getVectorAddThroughput()
{
	double average_time = 0.0;
	float time_elapsed;
	int number_threads = 0;
	int iterations = 2048;
	if (total_threads == 32 || MAX_OCCUPANCY) printf("\n-------------Vector-Addition-----------------\n"); 
	number_threads = total_threads / block_size;
	for (int j = 0; j < number_runs; j++) 
	{
		hipEventRecord(start, 0);
		kernelVectorAdd<<<block_size * number_multi_processors, number_threads>>>(device_A, device_B, operand);
		hipEventRecord(stop, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_elapsed, start, stop);
		average_time += time_elapsed;
	}
	print_results(average_time, number_runs, total_threads, iterations);
}

__global__ void kernelRemainder(TYPE* A, TYPE* B)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	TYPE a_val = A[i];
	TYPE b_val = B[i];
	repeat256(a_val %= b_val; b_val %= a_val;);
	A[i] = a_val;
	B[i] = b_val;
}

void getRemainderThroughput()
{
	double average_time = 0.0;
	float time_elapsed;
	int number_threads = 0;
	int iterations = 256;
	if (total_threads == 32 || MAX_OCCUPANCY) printf("\n-------------Remainder------------------\n"); 
	number_threads = total_threads / block_size;
	for (int j = 0; j < number_runs; j++) 
	{
		hipEventRecord(start, 0);
		kernelRemainder<<<block_size * number_multi_processors, number_threads>>>(device_A, device_B);
		hipEventRecord(stop, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_elapsed, start, stop);
		average_time += time_elapsed;
	}
	print_results(average_time, number_runs, total_threads, iterations);
}
#endif

#if DATATYPE == INT

__global__ void kernelAnd(int* A, int* B)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	TYPE a_val = A[i];
	TYPE b_val = B[i];
	repeat4096(a_val = a_val & b_val; b_val = a_val & b_val;);
	A[i] = a_val;
	B[i] = b_val;
}

void getAndThroughput()
{
	double average_time = 0.0;
	float time_elapsed;
	int number_threads = 0;
	int iterations = 4096;
	if (total_threads == 32 || MAX_OCCUPANCY) printf("\n-----------------AND-----------------------\n");
	number_threads = total_threads / block_size;
	for (int j = 0; j < number_runs; j++) 
	{
		hipEventRecord(start, 0);
		kernelAnd<<<block_size * number_multi_processors, number_threads>>>(device_A, device_B);
		hipEventRecord(stop, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_elapsed, start, stop);
		average_time += time_elapsed;
	}
	print_results(average_time, number_runs, total_threads, iterations);
}

__global__ void kernelOr(int* A, int* B)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	TYPE a_val = A[i];
	TYPE b_val = B[i];
	repeat4096(a_val = b_val | a_val; b_val = a_val | b_val;);
	A[i] = a_val;
	B[i] = b_val;
}

void getOrThroughput()
{
	double average_time = 0.0;
	float time_elapsed;
	int number_threads = 0;
	int iterations = 4096;
    	if (total_threads == 32 || MAX_OCCUPANCY) printf("\n-----------------OR------------------------\n"); 
	number_threads = total_threads / block_size;
	for (int j = 0; j < number_runs; j++) 
	{
		hipEventRecord(start, 0);
		kernelOr<<<block_size * number_multi_processors, number_threads>>>(device_A, device_B);
		hipEventRecord(stop, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_elapsed, start, stop);
		average_time += time_elapsed;
	}
	print_results(average_time, number_runs, total_threads, iterations);
}

__global__ void kernelXor(int* A, int* B)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	TYPE a_val = A[i];
	TYPE b_val = B[i];
	repeat4096(a_val = b_val ^ a_val; b_val = a_val ^ b_val;);
	A[i] = a_val;
	B[i] = b_val;
}

void getXorThroughput()
{
	double average_time = 0.0;
	float time_elapsed;
	int number_threads = 0;
	int iterations = 4096;
	if (total_threads == 32 || MAX_OCCUPANCY) printf("\n-----------------XOR-----------------------\n");
	number_threads = total_threads / block_size;
	for (int j = 0; j < number_runs; j++) 
	{
		hipEventRecord(start, 0);
		kernelXor<<<block_size * number_multi_processors, number_threads>>>(device_A, device_B);
		hipEventRecord(stop, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_elapsed, start, stop);
		average_time += time_elapsed;
	}
	print_results(average_time, number_runs, total_threads, iterations);
}

__global__ void kernelShl(int* A, int* B)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	TYPE a_val = A[i];
	TYPE b_val = B[i];
	repeat4096(a_val = b_val << a_val; b_val = a_val << b_val;);
	A[i] = a_val;
	B[i] = b_val;
}

void getShlThroughput()
{
	double average_time = 0.0;
	float time_elapsed;
	int number_threads = 0;
	int iterations = 4096;
	if (total_threads == 32 || MAX_OCCUPANCY) printf("\n--------------Shift-Left-------------------\n");
	number_threads = total_threads / block_size;
	for (int j = 0; j < number_runs; j++) 
	{
		hipEventRecord(start, 0);
		kernelShl<<<block_size * number_multi_processors, number_threads>>>(device_A, device_B);
		hipEventRecord(stop, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_elapsed, start, stop);
		average_time += time_elapsed;
	}
	print_results(average_time, number_runs, total_threads, iterations);
}

__global__ void kernelShr(int* A, int* B)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	TYPE a_val = A[i];
	TYPE b_val = B[i];
	repeat4096(a_val = b_val >> a_val; b_val = a_val >> b_val;);
	A[i] = a_val;
	B[i] = b_val;
}

void getShrThroughput()
{
	double average_time = 0.0;
	float time_elapsed;
	int number_threads = 0;
	int iterations = 4096;
	if (total_threads == 32 || MAX_OCCUPANCY) printf("\n--------------Shift-Right-------------------\n");
	number_threads = total_threads / block_size;
	for (int j = 0; j < number_runs; j++) 
	{
		hipEventRecord(start, 0);
		kernelShr<<<block_size * number_multi_processors, number_threads>>>(device_A, device_B);
		hipEventRecord(stop, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&time_elapsed, start, stop);
		average_time += time_elapsed;
	}
	print_results(average_time, number_runs, total_threads, iterations);
}

#endif

int main(int argc, char **argv)
{
	size_t array_size = N * sizeof(TYPE);
	hipEventCreate(&start);
	hipEventCreate(&stop);

	host_A = (TYPE *) malloc(array_size);
	host_B = (TYPE *) malloc(array_size);	
	
	if (host_A == NULL || host_B == NULL)
		exit(1);

	//Initilize arrays
	for (int i = 0; i < N; i++) 
	{
		host_A[i] = i * 1000;
		host_B[i] = i * 100;
	}

	hipMalloc((void**) &device_A, array_size);
	hipMalloc((void**) &device_B, array_size);

	//Copy values from host to device arrays
	hipMemcpy(device_A, host_A, array_size, hipMemcpyHostToDevice);
	hipMemcpy(device_B, host_B, array_size, hipMemcpyHostToDevice);
	
	switch(DATATYPE)
	{
		case INT: printf("**************Integer********************\n"); break;
		case UINT: printf("***********Unsigned-Integer***************\n"); break;
		case FLOAT: printf("**************Float********************\n"); break;
		case DOUBLE: printf("**************Double********************\n"); break;
	}
	
#if (MAX_OCCUPANCY)
	total_threads = max_warps * threads_per_warp;
	for (int k = 0; k < 12; k++) 
	{
		switch(k) 
		{
			case 0:	getAddThroughput();	break;
			case 1: getSubtractThroughput(); break;
			case 2: getMultiplyThroughput(); break;
			case 3: getDivideThroughput(); break;
			case 4: getMultiplyAddThroughput(); break;
#if DATATYPE == INT || DATATYPE == UINT
			case 5: getVectorAddThroughput(); break;
			case 6: getRemainderThroughput(); break;
#endif
#if DATATYPE == INT
			case 7: getAndThroughput(); break;
			case 8: getOrThroughput(); break;
			case 9: getXorThroughput(); break;
			case 10: getShlThroughput(); break;
			case 11: getShrThroughput(); break;
#endif
		}
	}
#endif
#if (!MAX_OCCUPANCY)
	for (int k = 0; k < 12; k++) {			
		for (int i = 1; i <= max_warps; i++) 
		{
			total_threads = i * threads_per_warp;
			switch(k) 
			{
				case 0:	getAddThroughput();	break;
				case 1: getSubtractThroughput(); break;
				case 2: getMultiplyThroughput(); break;
				case 3: getDivideThroughput(); break;
				case 4: getMultiplyAddThroughput(); break;
#if DATATYPE == INT || DATATYPE == UINT
				case 5: getVectorAddThroughput(); break;
				case 6: getRemainderThroughput(); break;
#endif
#if DATATYPE == INT
				case 7: getAndThroughput(); break;
				case 8: getOrThroughput(); break;
				case 9: getXorThroughput(); break;
				case 10: getShlThroughput(); break;
				case 11: getShrThroughput(); break;
#endif
			}
		}
	}
#endif
	
	hipFree(device_A);
	hipFree(device_B);
	free(host_A);
	free(host_B);
	
	return 0;
}
